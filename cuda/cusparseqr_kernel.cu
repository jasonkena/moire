#include <torch/extension.h>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#include <hipsparse.h>
#include <hipsolver.h>
//https://forums.developer.nvidia.com/t/cusolver-sparse-cusolverspdcsrlsvqr-error/38214
//https://stackoverflow.com/questions/31840341/solving-general-sparse-linear-systems-in-cuda

#define sparseErrchk(ans) { sparseAssert((ans), __FILE__, __LINE__); }
void sparseAssert(hipsparseStatus_t code, const char *file, int line, bool abort = true)
{
    if (code != HIPSPARSE_STATUS_SUCCESS)
    {
        fprintf(stderr, "GPUassert: %s %s %d\n", hipsparseGetErrorString(code), file, line);
        if (abort) { exit(code); }
    }
}


const char* cusolverGetErrorString(hipsolverStatus_t error);
#define solverErrchk(ans) { solverAssert((ans), __FILE__, __LINE__); }
void solverAssert(hipsolverStatus_t code, const char *file, int line, bool abort = true)
{
    if (code != HIPSOLVER_STATUS_SUCCESS)
    {
        fprintf(stderr, "GPUassert: %s %s %d\n", cusolverGetErrorString(code), file, line);
        if (abort) { exit(code); }
    }
}

const char* cusolverGetErrorString(hipsolverStatus_t error)
{
    switch (error)
    {
    case HIPSOLVER_STATUS_SUCCESS:
        return "CUSOLVER_SUCCESS";

    case HIPSOLVER_STATUS_NOT_INITIALIZED:
        return "HIPSOLVER_STATUS_NOT_INITIALIZED";

    case HIPSOLVER_STATUS_ALLOC_FAILED:
        return "HIPSOLVER_STATUS_ALLOC_FAILED";

    case HIPSOLVER_STATUS_INVALID_VALUE:
        return "HIPSOLVER_STATUS_INVALID_VALUE";

    case HIPSOLVER_STATUS_ARCH_MISMATCH:
        return "HIPSOLVER_STATUS_ARCH_MISMATCH";

    case HIPSOLVER_STATUS_EXECUTION_FAILED:
        return "HIPSOLVER_STATUS_EXECUTION_FAILED";

    case HIPSOLVER_STATUS_INTERNAL_ERROR:
        return "HIPSOLVER_STATUS_INTERNAL_ERROR";

    case HIPSOLVER_STATUS_MATRIX_TYPE_NOT_SUPPORTED:
        return "HIPSOLVER_STATUS_MATRIX_TYPE_NOT_SUPPORTED";

    }

    return "<unknown>";
}


int solve_cuda(int nnz, int m, double tol, double *dcooVal, int *dcooColInd, int *dcooRowInd, int *dcsrRowPtr, double *db, double *dx) {
  // --- create library handles:
  hipsolverSpHandle_t cusolver_handle;
  solverErrchk(hipsolverSpCreate(&cusolver_handle));

  hipsparseHandle_t cusparse_handle;
  sparseErrchk(hipsparseCreate(&cusparse_handle));

  // --- prepare solving and copy to GPU:
  int reorder = 0;
  int singularity = 0;

  // create matrix descriptor
  hipsparseMatDescr_t descrA;
  sparseErrchk(hipsparseCreateMatDescr(&descrA));
  hipsparseSetMatIndexBase(descrA, HIPSPARSE_INDEX_BASE_ZERO);
  hipsparseSetMatType(descrA, HIPSPARSE_MATRIX_TYPE_GENERAL);

  hipDeviceSynchronize();
  // convert COO to CSR
  sparseErrchk(hipsparseXcoo2csr(cusparse_handle,
                 dcooRowInd,
                 nnz,
                 m,
                 dcsrRowPtr,
                 HIPSPARSE_INDEX_BASE_ZERO));

  hipDeviceSynchronize();

  // solve the system
  solverErrchk(hipsolverSpDcsrlsvqr(cusolver_handle, m, nnz, descrA, dcooVal,
                                dcsrRowPtr, dcooColInd, db, tol, reorder, dx,
                                &singularity));

  hipDeviceSynchronize();

  sparseErrchk(hipsparseDestroy(cusparse_handle));
  solverErrchk(hipsolverSpDestroy(cusolver_handle));

  return singularity;
}


// write info about memory requirements of the qr-decomposition to stdout
void get_memInfo(int nnz, int m, double tol, double *csrVal, int *csrColInd,
                 int *csrRowPtr, double *b, double *x) {

    int* dCol, *dRow;
    double* dVal;
    hipError_t error;

    //allocate device memory, copy H2D
    hipMalloc((void**)&dCol, sizeof(int)*nnz);
    hipMalloc((void**)&dRow, sizeof(int)*(m+1));
    hipMalloc((void**)&dVal, sizeof(double)*nnz);
    hipMemcpy(dCol, csrColInd, sizeof(int)*nnz,    hipMemcpyHostToDevice);
    hipMemcpy(dRow, csrRowPtr, sizeof(int)*(m+1),  hipMemcpyHostToDevice);
    hipMemcpy(dVal, csrVal, sizeof(double)*nnz,  hipMemcpyHostToDevice);

    error = hipGetLastError();
    std::cout << "Error status after hipMemcpy in getmemInfo: " << error << std::endl;

    //create and initialize library handles
    hipsolverSpHandle_t cusolver_handle;
    hipsparseHandle_t cusparse_handle;
    hipsolverStatus_t cusolver_status;
    hipsparseStatus_t cusparse_status;
    cusparse_status = hipsparseCreate(&cusparse_handle);
    std::cout << "status hipsparseCreate: " << cusparse_status << std::endl;
    cusolver_status = hipsolverSpCreate(&cusolver_handle);
    std::cout << "status hipsolverSpCreate: " << cusolver_status << std::endl;

    //create CsrqrInfo
    csrqrInfo_t info;
    cusolver_status = cusolverSpCreateCsrqrInfo(&info);
    std::cout << "status cusolverSpCrateCsrqrInfo: " << cusolver_status << std::endl;

    //create mat descriptor
    hipsparseMatDescr_t descrA;
    cusparse_status = hipsparseCreateMatDescr(&descrA);
    cusparse_status = hipsparseSetMatType(descrA, HIPSPARSE_MATRIX_TYPE_GENERAL);
    std::cout << "status cusparse createMatDescr: " << cusparse_status << std::endl;
    hipDeviceSynchronize();
    //call SpDcsrqrAnalysisBatched.
    cusolver_status = cusolverSpXcsrqrAnalysisBatched(cusolver_handle,
                                                      m,
                                                      m,
                                                      nnz,
                                                      descrA,
                                                      dRow,
                                                      dCol,
                                                      info);
    std::cout << "status cusolverSpDcsrqrAnalysisBatched: " << cusolver_status << std::endl;

    //get the buffer size via BufferInfoBatched
    int batchsize = 1;
    size_t internalDataInBytes = 99;
    size_t workspaceInBytes = 99;
    cusolver_status = cusolverSpDcsrqrBufferInfoBatched(cusolver_handle,
                                                        m,
                                                        m,
                                                        nnz,
                                                        descrA,
                                                        dVal,
                                                        dRow,
                                                        dCol,
                                                        batchsize,
                                                        info,
                                                        &internalDataInBytes,
                                                        &workspaceInBytes);

    std::cout << "status cusolverSpDcsrqrBufferInfoBatched: " << cusolver_status << std::endl;
    std::cout << "internalbuffer(Bytes): " << internalDataInBytes << std::endl;
    std::cout << "workspace(Bytes): " << workspaceInBytes << std::endl;

    //destroy stuff
    cusolver_status = cusolverSpDestroyCsrqrInfo(info);
    std::cout << "status cusolverSpDestroyCsrqrInfo: " << cusolver_status << std::endl;

    cusparse_status = hipsparseDestroy(cusparse_handle);
    std::cout << "status hipsparseDestroy: " << cusparse_status << std::endl;
    cusolver_status = hipsolverSpDestroy(cusolver_handle);
    std::cout << "status hipsolverSpDestroy: " << cusolver_status << std::endl;
    hipFree(dCol);
    hipFree(dRow);
    hipFree(dVal);
}
